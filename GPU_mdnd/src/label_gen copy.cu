
#include <hip/hip_runtime.h>
// #include "label/gen_label.cuh"

// // 通过 hashtable 的快速查询
// __device__ int query_dis_by_hash_table (int u, int v, cuda_hashTable_v2<weight_type> *H, cuda_vector_v2<hub_type> *L, int hop_now, int hop_cst) {
//     int min_dis = 1e9;

//     int block_num = L->blocks_num;
//     int cnt = 0;
//     for (int i = 0; i < block_num; ++i) {
//         int block_id = L->block_idx_array[i];
//         // __threadfence_system();
//         int block_siz = L->pool->get_block_size(block_id);
//         // __threadfence_system();
//         for (int j = 0; j < block_siz; ++j) {
//             // hub_type* x = L->pool->get_node(block_id, j);
//             hub_type *x = &(L->pool->blocks_pool[block_id].data[j]);
//             // __threadfence_system();
//             for (int k = hop_now - x->hop; k >= 0; --k) {
//                 // int ddis = x->distance + H->get(x->hub_vertex, k, hop_cst);
//                 // if (ddis < min_dis){
//                 //     min_dis = ddis;
//                 //     break;
//                 // }
//                 min_dis = min(min_dis, x->distance + H->get(x->hub_vertex, k, hop_cst));
//                 // __threadfence_system();
//             }
//             cnt ++;
//             if (cnt >= L->last_size) break;
//         }
//     }
//     return min_dis;

// }

// // 动态并行加速查询
// // u, v, d_size, d_has, d, label, hop, hop_cst;
// // (node_id, d->current_size, das, d, has, L_gpu, t1, hop_now, hop_cst)
// __global__ void query_parallel (int sv, int st, int ed, int sz, cuda_hashTable_v2<weight_type> *das, int *d,
// cuda_hashTable_v2<weight_type> *has, cuda_vector_v2<hub_type> *L_gpu, cuda_vector_v2<T_item> *t1, int hop_now, int hop_cst) {
//     int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
//     if (tid < 0 || tid >= sz) {
//         return;
//     }

//     // 获取 D 队列元素
//     int v = d[st + tid];
//     weight_type dv = das->get(v);
//     weight_type q_dis = query_dis_by_hash_table(sv, v, has, L_gpu + v, hop_now + 1, hop_cst);
    
//     if (dv < q_dis) {
//         // 添加标签并压入 T 队列
//         L_gpu[v].push_back({sv, hop_now + 1, dv});
//         t1->push_back({v, dv});
//     }
//     // __syncthreads();

//     das->modify(v, 1e9);

// }

// // 初始化 T
// __global__ void init_T (int V, cuda_vector_v2<T_item> *T, cuda_vector_v2<hub_type> *L_gpu) {
//     int tid = blockIdx.x * blockDim.x + threadIdx.x;
//     if (tid < V) {
//         // target_vertex, distance
//         T[tid].push_back({tid, 0});
//         L_gpu[tid].push_back({tid, 0, 0});
//         L_gpu[tid].last_size = 1;
//     }
// }

// // 清空 T
// __global__ void clear_T (int V, cuda_vector_v2<T_item> *T, cuda_vector_v2<hub_type> *L_gpu) {
//     int tid = blockIdx.x * blockDim.x + threadIdx.x;
//     if (tid < V) {
//         T[tid].init(V, tid);
//         L_gpu[tid].last_size = L_gpu[tid].current_size;
//     }
// }

// // 索引生成过程，朴素的并行
// __global__ void gen_label_hsdl (int V, int thread_num, int hop_cst, int hop_now, int* out_pointer, int* out_edge, int* out_edge_weight,
//             cuda_vector_v2<hub_type> *L_gpu, cuda_hashTable_v2<weight_type> *Has, cuda_vector_v2<T_item> *T0, cuda_vector_v2<T_item> *T1) {
    
//     // 线程id
//     int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
//     if (tid < 0 || tid >= thread_num) {
//         return;
//     }

//     // hash table
//     cuda_hashTable_v2<weight_type> *has = (Has + tid);

//     for (int node_id = tid; node_id < V; node_id += thread_num) {

//         // node_id 的 T 队列
//         cuda_vector_v2<T_item> *t0 = (T0 + node_id);
//         cuda_vector_v2<T_item> *t1 = (T1 + node_id);

//         // node_id 的 label
//         cuda_vector_v2<hub_type> *L = (L_gpu + node_id);

//         // 初始化 hashtable，就是遍历 label 集合并一一修改在 hashtable 中的值
//         for (int i = 0; i < L->blocks_num; ++i) {
//             int block_id = L->block_idx_array[i];
//             int block_siz = L->pool->get_block_size(block_id);
//             for (int j = 0; j < block_siz; ++j) {
//                 hub_type* x = L->pool->get_node(block_id, j);
//                 has->modify(x->hub_vertex, x->hop, hop_cst, x->distance);
//             }
//         }

//         // 遍历 T 队列
//         for (int i = 0; i < t0->blocks_num; ++i) {
//             int block_id = t0->block_idx_array[i];
//             int block_siz = t0->pool->get_block_size(block_id);
//             for (int j = 0; j < block_siz; ++j) {

//                 // 获取 T 队列元素
//                 T_item *x = t0->pool->get_node(block_id, j);

//                 // sv 为起点, ev 为遍历到的点, dis 为距离，hop 为跳数
//                 int sv = node_id, ev = x->vertex, h = hop_now;
//                 weight_type dis = x->distance;

//                 // 遍历节点 ev 并扩展
//                 for (int k = out_pointer[ev]; k < out_pointer[ev + 1]; ++k) {
//                     int v = out_edge[k];
                    
//                     // rank pruning，并且同一个点也不能算。
//                     if (sv >= v) continue;

//                     // h 为现在这些标签的跳数， h + 1为现在要添加的标签跳数
//                     int dv = dis + out_edge_weight[k];
//                     weight_type q_dis = query_dis_by_hash_table(sv, v, Has + tid, L_gpu + v, h + 1, hop_cst);
                    
//                     if (dv < q_dis) {
//                         // 添加标签并压入 T 队列
//                         L_gpu[v].push_back({sv, h + 1, dv});
//                         t1->push_back({v, dv});
//                     }

//                 }
//             }
//         }

//         // 改回 hashtable
//         for (int i = 0; i < L->blocks_num; ++i) {
//             int block_id = L->block_idx_array[i];
//             int block_siz = L->pool->get_block_size(block_id);
//             for (int j = 0; j < block_siz; ++j) {
//                 hub_type* x = L->pool->get_node(block_id, j);
//                 has->modify(x->hub_vertex, x->hop, hop_cst, 1e9);
//             }
//         }
//     }
// }

// // 索引生成过程_v2，加入了 D 队列优化，无冗余
// __global__ void gen_label_hsdl_v2 (int V, int thread_num, int hop_cst, int hop_now, int* out_pointer, int* out_edge, int* out_edge_weight,
//             cuda_vector_v2<hub_type> *L_gpu, cuda_hashTable_v2<weight_type> *Has, cuda_hashTable_v2<weight_type> *Das,
//             cuda_vector_v2<T_item> *T0, cuda_vector_v2<T_item> *T1, int *d) {
    
//     // 线程id
//     int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
//     if (tid < 0 || tid >= thread_num) {
//         return;
//     }

//     // hash table
//     cuda_hashTable_v2<weight_type> *has = (Has + tid);
//     cuda_hashTable_v2<weight_type> *das = (Das + tid);
//     int d_start = tid * V, d_end = d_start;
//     int block_id, block_siz;

//     for (int node_id = tid; node_id < V; node_id += thread_num) {
        
//         // node_id 的 T 队列
//         cuda_vector_v2<T_item> *t0 = (T0 + node_id);
//         cuda_vector_v2<T_item> *t1 = (T1 + node_id);

//         // node_id 的 label
//         cuda_vector_v2<hub_type> *L = (L_gpu + node_id);

//         // 初始化 hashtable，就是遍历 label 集合并一一修改在 hashtable 中的值
//         int cnt = 0;
//         for (int i = 0; i < L->blocks_num; ++i) {
//             block_id = L->block_idx_array[i];
//             block_siz = L->pool->get_block_size(block_id);
//             for (int j = 0; j < block_siz; ++j) {
//                 hub_type* x = L->pool->get_node(block_id, j);
//                 has->modify(x->hub_vertex, x->hop, hop_cst, x->distance);
//                 cnt ++;
//                 // if (cnt >= L->last_size) break;
//             }
//         }

//         // 遍历 T 队列，并生成 D 队列
//         for (int i = 0; i < t0->blocks_num; ++i) {
//             block_id = t0->block_idx_array[i];
//             block_siz = t0->pool->get_block_size(block_id);
//             for (int j = 0; j < block_siz; ++j) {

//                 // 获取 T 队列元素
//                 T_item *x = t0->pool->get_node(block_id, j);

//                 // sv 为起点, ev 为遍历到的点, dis 为距离，hop 为跳数
//                 int sv = node_id, ev = x->vertex, h = hop_now;
//                 weight_type dis = x->distance;

//                 // 遍历节点 ev 并扩展
//                 for (int k = out_pointer[ev]; k < out_pointer[ev + 1]; ++k) {
//                     int v = out_edge[k];
                    
//                     // rank pruning，并且同一个点也不能算。
//                     if (sv >= v) continue;

//                     // h 为现在这些标签的跳数， h + 1为现在要添加的标签跳数
//                     int dv = dis + out_edge_weight[k];

//                     // 判断生成 D 队列
//                     weight_type d_hash = das->get(v);
//                     if (d_hash == 1e9) {
//                         d[d_end ++] = v;
//                         das->modify(v, dv);
//                     }else{
//                         if (d_hash > dv) {
//                             das->modify(v, dv);
//                         }
//                     }

//                 }
//             }
//         }

//         // 遍历 D 队列
//         for (int i = d_start; i < d_end; ++i) {
//             int sv = node_id, v = d[i], h = hop_now;
//             weight_type dv = das->get(v);
//             weight_type q_dis = query_dis_by_hash_table(sv, v, has, L_gpu + v, h + 1, hop_cst);
            
//             if (dv < q_dis) {
//                 // 添加标签并压入 T 队列
//                 L_gpu[v].push_back({sv, h + 1, dv});
//                 t1->push_back({v, dv});
//             }

//             das->modify(v, 1e9);
//         }

//         // 改回 hashtable
//         cnt = 0;
//         for (int i = 0; i < L->blocks_num; ++i) {
//             block_id = L->block_idx_array[i];
//             block_siz = L->pool->get_block_size(block_id);
//             for (int j = 0; j < block_siz; ++j) {
//                 hub_type* x = L->pool->get_node(block_id, j);
//                 has->modify(x->hub_vertex, x->hop, hop_cst, 1e9);
//                 cnt ++;
//                 // if (cnt >= L->last_size) break;
//             }
//         }
//     }
// }

// // 索引生成过程_v3，加入了 D 队列优化，实现了 D 队列遍历的并行，无冗余
// __global__ void gen_label_hsdl_v3 (int V, int thread_num, int hop_cst, int hop_now, int* out_pointer, int* out_edge, int* out_edge_weight,
//             cuda_vector_v2<hub_type> *L_gpu, cuda_hashTable_v2<weight_type> *Has, cuda_hashTable_v2<weight_type> *Das,
//             cuda_vector_v2<T_item> *T0, cuda_vector_v2<T_item> *T1, int *d,
//             clock_t *timer_hash1, clock_t *timer_gett, clock_t *timer_query, clock_t *timer_hash2) {
    
//     // 线程id
//     int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
//     if (tid < 0 || tid >= thread_num) {
//         return;
//     }

//     // hash table
//     cuda_hashTable_v2<weight_type> *has = (Has + tid);
//     cuda_hashTable_v2<weight_type> *das = (Das + tid);
//     int d_start, d_end;
//     int block_id, block_siz;
    
//     for (int node_id = tid; node_id < V; node_id += thread_num) {
        
//         d_start = tid * V;
//         d_end = d_start;

//         // node_id 的 T 队列
//         cuda_vector_v2<T_item> *t0 = (T0 + node_id);
//         cuda_vector_v2<T_item> *t1 = (T1 + node_id);

//         // node_id 的 label
//         cuda_vector_v2<hub_type> *L = (L_gpu + node_id);
        
//         timer_hash1[tid] += clock() / 1000;
//         // 初始化 hashtable，就是遍历 label 集合并一一修改在 hashtable 中的值
//         int cnt = 0;
//         for (int i = 0; i < L->blocks_num; ++i) {
//             block_id = L->block_idx_array[i];
//             block_siz = L->pool->get_block_size(block_id);
//             for (int j = 0; j < block_siz; ++j) {
//                 hub_type *x = &(L->pool->blocks_pool[block_id].data[j]);
//                 has->modify(x->hub_vertex, x->hop, hop_cst, x->distance);
//                 cnt ++;
//                 if (cnt >= L->last_size) break;
//             }
//         }
//         timer_hash1[tid + thread_num] += clock() / 1000;

//         timer_gett[tid] += clock() / 1000;
//         // 遍历 T 队列，并生成 D 队列
//         for (int i = 0; i < t0->blocks_num; ++i) {
//             block_id = t0->block_idx_array[i];
//             block_siz = t0->pool->get_block_size(block_id);
//             for (int j = 0; j < block_siz; ++j) {

//                 // 获取 T 队列元素
//                 // T_item *x = t0->pool->get_node(block_id, j);
//                 T_item *x = &(t0->pool->blocks_pool[block_id].data[j]);

//                 // sv 为起点, ev 为遍历到的点, dis 为距离，hop 为跳数
//                 int sv = node_id, ev = x->vertex, h = hop_now;
//                 weight_type dis = x->distance;

//                 // 遍历节点 ev 并扩展
//                 for (int k = out_pointer[ev]; k < out_pointer[ev + 1]; ++k) {
//                     int v = out_edge[k];
                    
//                     // rank pruning，并且同一个点也不能算。
//                     if (sv >= v) continue;

//                     // h 为现在这些标签的跳数， h + 1为现在要添加的标签跳数
//                     int dv = dis + out_edge_weight[k];

//                     // 判断生成 D 队列
//                     weight_type d_hash = das->get(v);
//                     if (d_hash == 1e9) {
//                         d[d_end ++] = v;
//                         das->modify(v, dv);
//                     }else{
//                         if (d_hash > dv) {
//                             das->modify(v, dv);
//                         }
//                     }

//                 }
//             }
//         }
//         timer_gett[tid + thread_num] += clock() / 1000;

//         // u, v, d_size, hash, label, t, hop, hop_cst;
//         timer_query[tid] += clock() / 1000;
//         query_parallel <<< (d_end - d_start + 1023) / 1024, 1024 >>>
//         (node_id, d_start, d_end, d_end - d_start, das, &d[0], has, L_gpu, t1, hop_now, hop_cst);
//         cudaDeviceSynchronize();
//         timer_query[tid + thread_num] += clock() / 1000;

//         // 改回 hashtable
//         timer_hash2[tid] += clock() / 1000;
//         cnt = 0;
//         for (int i = 0; i < L->blocks_num; ++i) {
//             block_id = L->block_idx_array[i];
//             block_siz = L->pool->get_block_size(block_id);
//             for (int j = 0; j < block_siz; ++j) {
//                 hub_type *x = &(L->pool->blocks_pool[block_id].data[j]);
//                 has->modify(x->hub_vertex, x->hop, hop_cst, 1e9);
//                 cnt ++;
//                 if (cnt >= L->last_size) break;
//             }
//         }
//         timer_hash2[tid + thread_num] += clock() / 1000;
//     }
    
// }

// __global__ void add_timer (clock_t* tot, clock_t *t, int thread_num) {
//     for (int i = 0; i < thread_num; ++i) {
//         (*tot) += (t[i + thread_num] - t[i]) / 1000;
//     }
//     printf("t: %lld\n", (long long)(*tot));
// }

// // 生成 label 的过程
// void label_gen (CSR_graph<weight_type>& input_graph, hop_constrained_case_info_v2 *info, int hop_cst, vector<vector<hub_type> >&L) {

//     int V = input_graph.OUTs_Neighbor_start_pointers.size() - 1;
//     int E = input_graph.OUTs_Edges.size();
//     int* out_edge = input_graph.out_edge;
//     int* out_edge_weight = input_graph.out_edge_weight;
//     int* out_pointer = input_graph.out_pointer;

//     int thread_num = 1000;

//     int dimGrid = 1, dimGrid_V, dimBlock = 1;
//     dimGrid_V = (V + dimBlock - 1) / dimBlock;
//     dimGrid = (thread_num + dimBlock - 1) / dimBlock;

//     printf("V, E: %d, %d\n", V, E);

//     // 准备 info
//     info->init(V, V * V * (hop_cst + 1), hop_cst);
//     printf("init case_info success\n");

//     // 准备 L_hashTable
//     cuda_hashTable_v2<weight_type> *L_hash;
//     cudaMallocManaged(&L_hash, thread_num * sizeof(cuda_hashTable_v2<weight_type>));
//     cudaDeviceSynchronize();
//     for (int i = 0; i < thread_num; i++) {
//         new (L_hash + i) cuda_hashTable_v2 <weight_type> (V * (hop_cst + 1));
//     }

//     // 准备 D_hashTable
//     cuda_hashTable_v2<weight_type> *D_hash;
//     cudaMallocManaged(&D_hash, thread_num * sizeof(cuda_hashTable_v2<weight_type>));
//     cudaDeviceSynchronize();
//     for (int i = 0; i < thread_num; i++) {
//         new (D_hash + i) cuda_hashTable_v2 <weight_type> (V);
//     }
//     printf("init hash_table success\n");

//     // 准备 D_vector
//     int *D_vector;
//     cudaMallocManaged(&D_vector, thread_num * V * sizeof(int));

//     // 编号越小的点，rank 越高
//     // for (int i = 0; i < V; i ++){
//     //     printf("degree %d, %d\n", i, out_pointer[i + 1] - out_pointer[i]);
//     // }

//     // 同步，保证数据初始化完成
//     cudaDeviceSynchronize();

//     // 计时 label gen 中的耗时
//     clock_t *timer_hash1, *timer_gett, *timer_query, *timer_hash2;
//     clock_t *timer_hash1_tot, *timer_gett_tot, *timer_query_tot, *timer_hash2_tot;
//     cudaMallocManaged(&timer_hash1, thread_num * 2 * sizeof(clock_t));
//     cudaMallocManaged(&timer_hash1_tot, sizeof(clock_t));
//     cudaMallocManaged(&timer_gett, thread_num * 2 * sizeof(clock_t));
//     cudaMallocManaged(&timer_gett_tot, sizeof(clock_t));
//     cudaMallocManaged(&timer_query, thread_num * 2 * sizeof(clock_t));
//     cudaMallocManaged(&timer_query_tot, sizeof(clock_t));
//     cudaMallocManaged(&timer_hash2, thread_num * 2 * sizeof(clock_t));
//     cudaMallocManaged(&timer_hash2_tot, sizeof(clock_t));
    
//     // 测试 cuda_vector 和 cuda_hash 的部分
//     // test_mmpool(V, thread_num, 2, info, L_hash);
    
//     init_T <<<dimGrid_V, dimBlock>>> (V, info->T0, info->L_cuda);
//     cudaDeviceSynchronize();

//     // 计时
//     cudaEvent_t start, stop;
//     float elapsedTime = 0.0;
//     cudaEventCreate(&start);
//     cudaEventCreate(&stop);
//     cudaEventRecord(start, 0);
    
//     // 辅助变量，不方便直接探测 T 是否为空
//     int iter = 0;

//     while (1) {

//         if (iter++ >= hop_cst) break;
//         // iter = 1 -> 生成 跳数 1
//         // iter = 2 -> 生成 跳数 2
//         // iter = 3 -> 生成 跳数 3
//         // iter = 4 -> 生成 跳数 4
//         // iter = 5 -> 生成 跳数 5
        
//         printf("iteration_hop: %d\n", iter);

//         // 根据奇偶性，轮流使用 T0、T1，不需要交换指针
//         if (iter % 2 == 1) {
//             // gen_label_hsdl <<<dimGrid, dimBlock>>> (V, thread_num, hop_cst, iter - 1, out_pointer, out_edge, out_edge_weight,
//             // info->L_cuda, L_hash, info->T0, info->T1);
//             // gen_label_hsdl_v2 <<< dimGrid, dimBlock >>> (V, thread_num, hop_cst, iter - 1, out_pointer, out_edge, out_edge_weight,
//             // info->L_cuda, L_hash, D_hash, info->T0, info->T1, D_vector);
//             gen_label_hsdl_v3 <<< dimGrid, dimBlock >>> (V, thread_num, hop_cst, iter - 1, out_pointer, out_edge, out_edge_weight,
//             info->L_cuda, L_hash, D_hash, info->T0, info->T1, D_vector, timer_hash1, timer_gett, timer_query, timer_hash2);
//             cudaDeviceSynchronize();

//             // 清洗 T 数组
//             clear_T <<< dimGrid_V, dimBlock >>> (V, info->T0, info->L_cuda);
//         }else{
//             // gen_label_hsdl <<<dimGrid, dimBlock>>> (V, thread_num, hop_cst, iter - 1, out_pointer, out_edge, out_edge_weight,
//             // info->L_cuda, L_hash, info->T1, info->T0);
//             // gen_label_hsdl_v2 <<< dimGrid, dimBlock >>> (V, thread_num, hop_cst, iter - 1, out_pointer, out_edge, out_edge_weight,
//             // info->L_cuda, L_hash, D_hash, info->T1, info->T0, D_vector);
//             gen_label_hsdl_v3 <<< dimGrid, dimBlock >>> (V, thread_num, hop_cst, iter - 1, out_pointer, out_edge, out_edge_weight,
//             info->L_cuda, L_hash, D_hash, info->T1, info->T0, D_vector, timer_hash1, timer_gett, timer_query, timer_hash2);
//             cudaDeviceSynchronize();

//             // 清洗 T 数组
//             clear_T <<< dimGrid_V, dimBlock >>> (V, info->T1, info->L_cuda);
//         }
//         cudaDeviceSynchronize();
//         cudaEventRecord(stop, 0);
//         cudaEventSynchronize(stop);
//         cudaEventElapsedTime(&elapsedTime, start, stop);
//         printf("Time generation in hop %d : %.8lf s\n", iter, elapsedTime / 1000.0);

//     }
//     cudaError_t err;
//     err = cudaGetLastError(); // 检查内核内存申请错误
//     if (err != cudaSuccess) {
//         printf("!INIT CUDA ERROR: %s\n", cudaGetErrorString(err));
//     }
    
//     // timer record
//     cudaEventRecord(stop, 0);
//     cudaEventSynchronize(stop);
//     cudaEventElapsedTime(&elapsedTime, start, stop);
//     printf("Time generation: %.8f s\n", elapsedTime / 1000.0);
//     info->time_generate_labels = elapsedTime / 1000.0;
//     cudaEventDestroy(start);
//     cudaEventDestroy(stop);

//     add_timer <<< 1, 1 >>> (timer_hash1_tot, timer_hash1, thread_num);
//     add_timer <<< 1, 1 >>> (timer_gett_tot, timer_gett, thread_num);
//     add_timer <<< 1, 1 >>> (timer_query_tot, timer_query, thread_num);
//     add_timer <<< 1, 1 >>> (timer_hash2_tot, timer_hash2, thread_num);

//     // printf("hub, parent, hop, dis:\n");
//     auto beforeTime = std::chrono::steady_clock::now();
//     info->label_size = 0;
//     for (int v = 0; v < V; ++v) {
//         L[v].clear();
//         // printf("vertex %d\n", v);
//         for (int i = 0; i < info->L_cuda[v].blocks_num; ++i) {
//             int block_id = info->L_cuda[v].block_idx_array[i];
//             int block_siz = info->L_cuda[v].pool->get_block_size_host(block_id);
//             for (int j = 0; j < block_siz; ++j) {
//                 hub_type* x = info->L_cuda[v].pool->get_node_host(block_id, j);
//                 // printf("{%d, %d, %d, %d}, ", x->hub_vertex, x->parent_vertex, x->hop, x->distance);
//                 L[v].push_back({x->hub_vertex, x->hop, x->distance});
//                 // info->L_cpu[v].push_back({x->hub_vertex, x->hop, x->distance});
//                 info->label_size ++;
//             }
//         }
//         // printf("\n");
//     }
//     auto afterTime = std::chrono::steady_clock::now();
//     printf("time traverse labels: %.8lf\n", std::chrono::duration<double>(afterTime - beforeTime).count());

//     info->label_size = info->label_size / (double)V;
//     printf("average label size: %.8lf\n", info->label_size);
//     printf("Generation end!\n");

//     info->destroy_L_cuda();
    
//     for(int i = 0; i < thread_num; ++i){
//         L_hash[i].~cuda_hashTable_v2();
//     }
//     cudaFree(L_hash);
    
//     return;
// }