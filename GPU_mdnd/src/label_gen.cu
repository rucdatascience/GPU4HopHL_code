#include "hip/hip_runtime.h"
#include "label/gen_label.cuh"

// ͨ�� hashtable �Ŀ��ٲ�ѯ
__device__ int query_dis_by_hash_table (int u, int v, cuda_hashTable_v2<weight_type> *H, cuda_vector_v2<hub_type> *L, int hop_now, int hop_cst) {
    int min_dis = 1e9;

    for (int i = 0; i < L->blocks_num; ++i) {
        int block_id = L->block_idx_array[i];
        int block_siz = L->pool->get_block_size(block_id);
        for (int j = 0; j < block_siz; ++j) {
            hub_type* x = L->pool->get_node(block_id, j);
            for (int k = hop_now - x->hop; k >= 0; --k) {
                min_dis = min(min_dis, x->distance + H->get(x->hub_vertex, k, hop_cst));
            }
        }
    }
    return min_dis;

}

// ��ʼ�� T
__global__ void init_T (int V, cuda_vector_v2<T_item> *T, cuda_vector_v2<hub_type> *L_gpu) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < V) {
        // target_vertex, distance
        T[tid].push_back({tid, 0});
        L_gpu[tid].push_back({tid, 0, 0});
    }
}

// ��� T
__global__ void clear_T (int V, cuda_vector_v2<T_item> *T, cuda_vector_v2<T_item> *D) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < V) {
        T[tid].init(V, tid);
        D[tid].init(V, tid);
    }
}

// �������ɹ���
__global__ void gen_label_hsdl (int V, int thread_num, int hop_cst, int hop_now, int* out_pointer, int* out_edge, int* out_edge_weight,
            cuda_vector_v2<hub_type> *L_gpu, cuda_hashTable_v2<weight_type> *Has, cuda_vector_v2<T_item> *T0, cuda_vector_v2<T_item> *T1) {
    
    // �߳�id
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // hash table
    cuda_hashTable_v2<weight_type> *has = (Has + tid);

    for (int node_id = tid; node_id < V; node_id += thread_num) {

        // node_id �� T ����
        cuda_vector_v2<T_item> *t0 = (T0 + node_id);
        cuda_vector_v2<T_item> *t1 = (T1 + node_id);

        // node_id �� label
        cuda_vector_v2<hub_type> *L = (L_gpu + node_id);

        // ��ʼ�� hashtable�����Ǳ��� label ���ϲ�һһ�޸��� hashtable �е�ֵ
        for (int i = 0; i < L->blocks_num; ++i) {
            int block_id = L->block_idx_array[i];
            int block_siz = L->pool->get_block_size(block_id);
            for (int j = 0; j < block_siz; ++j) {
                hub_type* x = L->pool->get_node(block_id, j);
                has->modify(x->hub_vertex, x->hop, hop_cst, x->distance);
            }
        }

        // ���� T ����
        for (int i = 0; i < t0->blocks_num; ++i) {
            int block_id = t0->block_idx_array[i];
            int block_siz = t0->pool->get_block_size(block_id);
            for (int j = 0; j < block_siz; ++j) {

                // ��ȡ T ����Ԫ��
                T_item *x = t0->pool->get_node(block_id, j);

                // sv Ϊ���, ev Ϊ�������ĵ�, dis Ϊ���룬hop Ϊ����
                int sv = node_id, ev = x->vertex, h = hop_now;
                weight_type dis = x->distance;

                // �����ڵ� ev ����չ
                for (int k = out_pointer[ev]; k < out_pointer[ev + 1]; ++k) {
                    int v = out_edge[k];
                    
                    // rank pruning������ͬһ����Ҳ�����㡣
                    if (sv >= v) continue;

                    // h Ϊ������Щ��ǩ�������� h + 1Ϊ����Ҫ��ӵı�ǩ����
                    int dv = dis + out_edge_weight[k];
                    weight_type q_dis = query_dis_by_hash_table(sv, v, Has + tid, L_gpu + v, h + 1, hop_cst);
                    
                    if (dv < q_dis) {
                        // ��ӱ�ǩ��ѹ�� T ����
                        L_gpu[v].push_back({sv, h + 1, dv});
                        t1->push_back({v, dv});
                    }

                }
            }
        }

        // �Ļ� hashtable
        for (int i = 0; i < L->blocks_num; ++i) {
            int block_id = L->block_idx_array[i];
            int block_siz = L->pool->get_block_size(block_id);
            for (int j = 0; j < block_siz; ++j) {
                hub_type* x = L->pool->get_node(block_id, j);
                has->modify(x->hub_vertex, x->hop, hop_cst, 1e9);
            }
        }
    }
}

// �������ɹ���_v2
__global__ void gen_label_hsdl_v2 (int V, int thread_num, int hop_cst, int hop_now, int* out_pointer, int* out_edge, int* out_edge_weight,
            cuda_vector_v2<hub_type> *L_gpu, cuda_hashTable_v2<weight_type> *Has, cuda_hashTable_v2<weight_type> *Das,
            cuda_vector_v2<T_item> *T0, cuda_vector_v2<T_item> *T1, cuda_vector_v2<T_item> *D) {
    
    // �߳�id
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    
    // hash table
    cuda_hashTable_v2<weight_type> *has = (Has + tid);
    cuda_hashTable_v2<weight_type> *das = (Das + tid);

    for (int node_id = tid; node_id < V; node_id += thread_num) {

        // node_id �� T ����
        cuda_vector_v2<T_item> *t0 = (T0 + node_id);
        cuda_vector_v2<T_item> *t1 = (T1 + node_id);
        cuda_vector_v2<T_item> *d = (D + node_id);

        // node_id �� label
        cuda_vector_v2<hub_type> *L = (L_gpu + node_id);

        // ��ʼ�� hashtable�����Ǳ��� label ���ϲ�һһ�޸��� hashtable �е�ֵ
        for (int i = 0; i < L->blocks_num; ++i) {
            int block_id = L->block_idx_array[i];
            int block_siz = L->pool->get_block_size(block_id);
            for (int j = 0; j < block_siz; ++j) {
                hub_type* x = L->pool->get_node(block_id, j);
                has->modify(x->hub_vertex, x->hop, hop_cst, x->distance);
            }
        }

        // ���� T ���У������� D ����
        for (int i = 0; i < t0->blocks_num; ++i) {
            int block_id = t0->block_idx_array[i];
            int block_siz = t0->pool->get_block_size(block_id);
            for (int j = 0; j < block_siz; ++j) {

                // ��ȡ T ����Ԫ��
                T_item *x = t0->pool->get_node(block_id, j);

                // sv Ϊ���, ev Ϊ�������ĵ�, dis Ϊ���룬hop Ϊ����
                int sv = node_id, ev = x->vertex, h = hop_now;
                weight_type dis = x->distance;

                // �����ڵ� ev ����չ
                for (int k = out_pointer[ev]; k < out_pointer[ev + 1]; ++k) {
                    int v = out_edge[k];
                    
                    // rank pruning������ͬһ����Ҳ�����㡣
                    if (sv >= v) continue;

                    // h Ϊ������Щ��ǩ�������� h + 1Ϊ����Ҫ��ӵı�ǩ����
                    int dv = dis + out_edge_weight[k];

                    // �ж����� D ����
                    weight_type d_hash = das->get(v);
                    if (d_hash == 1e9) {
                        d->push_back({v, d_hash});
                        das->modify(v, dv);
                    }else{
                        if (d_hash > dv) {
                            das->modify(v, dv);
                        }
                    }

                }
            }
        }

        // ���� D ����
        for (int i = 0; i < d->blocks_num; ++i) {
            int block_id = d->block_idx_array[i];
            int block_siz = d->pool->get_block_size(block_id);
            for (int j = 0; j < block_siz; ++j) {

                // ��ȡ D ����Ԫ��
                T_item *x = d->pool->get_node(block_id, j);

                int sv = node_id, v = x->vertex, h = hop_now;
                weight_type dv = das->get(v);
                weight_type q_dis = query_dis_by_hash_table(sv, v, Has + tid, L_gpu + v, h + 1, hop_cst);
                
                if (dv < q_dis) {
                    // ��ӱ�ǩ��ѹ�� T ����
                    L_gpu[v].push_back({sv, h + 1, dv});
                    t1->push_back({v, dv});
                }

                das->modify(v, 1e9);
            }
        }

        // �Ļ� hashtable
        for (int i = 0; i < L->blocks_num; ++i) {
            int block_id = L->block_idx_array[i];
            int block_siz = L->pool->get_block_size(block_id);
            for (int j = 0; j < block_siz; ++j) {
                hub_type* x = L->pool->get_node(block_id, j);
                has->modify(x->hub_vertex, x->hop, hop_cst, 1e9);
            }
        }
    }
}

// ���� label �Ĺ���
void label_gen (CSR_graph<weight_type>& input_graph, hop_constrained_case_info_v2 *info, int hop_cst, vector<vector<hub_type> >&L) {
    
    int V = input_graph.OUTs_Neighbor_start_pointers.size() - 1;
    int E = input_graph.OUTs_Edges.size();
    int* out_edge = input_graph.out_edge;
    int* out_edge_weight = input_graph.out_edge_weight;
    int* out_pointer = input_graph.out_pointer;

    int dimGrid = 1, dimBlock = 1024;
    dimGrid = (V + dimBlock - 1) / dimBlock;

    int thread_num = 1000;

    printf("V, E: %d, %d\n", V, E);

    // ׼�� info
    info->init(V, V * V * hop_cst, hop_cst);
    printf("init case_info success\n");

    // ׼�� L_hashTable
    cuda_hashTable_v2<weight_type> *L_hash;
    hipMallocManaged(&L_hash, thread_num * sizeof(cuda_hashTable_v2<weight_type>));
    for (int i = 0; i < thread_num; i++) {
        new (L_hash + i) cuda_hashTable_v2 <weight_type> (V * (hop_cst + 1));
    }

    // ׼�� D_hashTable
    cuda_hashTable_v2<weight_type> *D_hash;
    hipMallocManaged(&D_hash, thread_num * sizeof(cuda_hashTable_v2<weight_type>));
    for (int i = 0; i < thread_num; i++) {
        new (D_hash + i) cuda_hashTable_v2 <weight_type> (V);
    }
    
    printf("init hash_table success\n");
    
    // ���ԽС�ĵ㣬rank Խ��
    // for (int i = 0; i < V; i ++){
    //     printf("degree %d, %d\n", i, out_pointer[i + 1] - out_pointer[i]);
    // }

    // ͬ������֤���ݳ�ʼ�����
    hipDeviceSynchronize();

    // ���� cuda_vector �� cuda_hash �Ĳ���
    // test_mmpool(V, thread_num, 3, info, L_hash)
    
    init_T <<<dimGrid, dimBlock>>> (V, info->T0, info->L_cuda);
    hipDeviceSynchronize();

    // ��ʱ
    hipEvent_t start, stop;
    float elapsedTime = 0.0;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    
    // ����������������ֱ��̽�� T �Ƿ�Ϊ��
    int iter = 0;

    while (1) {

        if (iter++ >= hop_cst) break;
        // iter = 1 -> ���� ���� 1
        // iter = 2 -> ���� ���� 2
        // iter = 3 -> ���� ���� 3
        // iter = 4 -> ���� ���� 4
        // iter = 5 -> ���� ���� 5
        
        printf("iteration_hop: %d\n", iter);

        // ������ż�ԣ�����ʹ�� T0��T1������Ҫ����ָ��
        if (iter % 2 == 1) {
            // gen_label_hsdl <<<1, thread_num>>> (V, thread_num, hop_cst, iter - 1, out_pointer, out_edge, out_edge_weight,
            // info->L_cuda, L_hash, info->T0, info->T1);
            gen_label_hsdl_v2 <<<1, thread_num>>> (V, thread_num, hop_cst, iter - 1, out_pointer, out_edge, out_edge_weight,
            info->L_cuda, L_hash, D_hash, info->T0, info->T1, info->D);
            hipDeviceSynchronize();

            // ��ϴ T ����
            clear_T <<<dimGrid, dimBlock>>> (V, info->T0, info->D);
        }else{
            // gen_label_hsdl <<<1, thread_num>>> (V, thread_num, hop_cst, iter - 1, out_pointer, out_edge, out_edge_weight,
            // info->L_cuda, L_hash, info->T1, info->T0);
            gen_label_hsdl_v2 <<<1, thread_num>>> (V, thread_num, hop_cst, iter - 1, out_pointer, out_edge, out_edge_weight,
            info->L_cuda, L_hash, D_hash, info->T1, info->T0, info->D);
            hipDeviceSynchronize();

            // ��ϴ T ����
            clear_T <<<dimGrid, dimBlock>>> (V, info->T1, info->D);
        }
        hipDeviceSynchronize();
        hipEventRecord(stop, 0);
        hipEventSynchronize(stop);
        hipEventElapsedTime(&elapsedTime, start, stop);
        printf("Time generation in hop %d : %.5lf s\n", iter, elapsedTime / 1000.0);

    }
    hipError_t err;
    err = hipGetLastError(); // ����ں��ڴ��������
    if (err != hipSuccess) {
        printf("!INIT CUDA ERROR: %s\n", hipGetErrorString(err));
    }
    
    // timer record
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("Time generation: %.5f s\n", elapsedTime / 1000.0);
    info->time_generate_labels = elapsedTime / 1000.0;
    hipEventDestroy(start);
    hipEventDestroy(stop);

    printf("hub, parent, hop, dis:\n");
    int mx_hop = 0;
    for (int v = 0; v < V; ++v) {
        // printf("vertex %d\n", v);
        for (int i = 0; i < info->L_cuda[v].blocks_num; ++i) {
            int block_id = info->L_cuda[v].block_idx_array[i];
            int block_siz = info->L_cuda[v].pool->get_block_size(block_id);
            for (int j = 0; j < block_siz; ++j) {
                hub_type* x = info->L_cuda[v].pool->get_node(block_id, j);
                // printf("{%d, %d, %d, %d}, ", x->hub_vertex, x->parent_vertex, x->hop, x->distance);
                L[v].push_back({x->hub_vertex, x->hop, x->distance});
                info->label_size ++;
                mx_hop = max(mx_hop, x->hop);
            }
        }
        // printf("\n");
    }
    printf("max hop: %d\n", mx_hop);
    info->label_size = info->label_size / (double)V;
    printf("average label size: %.5lf\n", info->label_size);
    printf("Generation end!\n");

    info->destroy_L_cuda();
    
    for(int i = 0; i < thread_num; ++i){
        L_hash[i].~cuda_hashTable_v2();
    }
    hipFree(L_hash);
    
    return;
}