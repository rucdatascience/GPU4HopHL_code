#include "hip/hip_runtime.h"
#include <iostream>
#include <hip/hip_runtime.h>
#include <HBPLL/gpu_clean.cuh>
#include <utility>

#define THREADS_PER_BLOCK 1024
#define clean_thread_num 1000

int *L2_pos_2;
int *L_size_2;
long long L_tot_2 = 0;
vector<long long> label_id;

// 64bits, hub_vertex 24bits, parent_vertex 24bits, hop 3bits, distance 10bits
inline __host__ __device__ int get_hub_vertex (long long x) {
    return (x >> 37);
}
inline __host__ __device__ int get_parent_vertex (long long x) {
    return (x >> 13) & ((1 << 24) - 1);
}
inline __host__ __device__ int get_hop (long long x) {
    return (x >> 10) & ((1 << 3) - 1);
}
inline __host__ __device__ int get_distance (long long x) {
    return (x) & ((1 << 10) - 1);
}
inline __host__ __device__ long long get_label (int hub_vertex, int parent_vertex, int hop, int distance) {
    return ((long long)hub_vertex << 37) | ((long long)parent_vertex << 13) | ((long long)hop << 10) | ((long long)distance);
}

inline bool operator < (hop_constrained_two_hop_label a, hop_constrained_two_hop_label b) {
    return a.hub_vertex < b.hub_vertex;
}
inline bool operator > (hop_constrained_two_hop_label a, hop_constrained_two_hop_label b) {
    return a.hub_vertex < b.hub_vertex;
}

inline bool cmp_LL(std::pair<long long, int> x, std::pair<long long, int> y) {
    int vx1 = get_hub_vertex(x.first);
    int vx2 = get_parent_vertex(x.first);
    int vy1 = get_hub_vertex(y.first);
    int vy2 = get_parent_vertex(y.first);
    return min(vx1, vx2) > min(vy1, vy2);
}

// get hash_table
__global__ void get_hash_v3 (int V, int hop_cst, int vid, int *in_L, long long *L, long long *L_start, long long *L_end, int *hash_array) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < 0 || tid >= L_end[vid] - L_start[vid]) {
        return;
    }
    long long LL = L[L_start[vid] + tid]; // get the label need to get hash
    int hub_vertex = get_hub_vertex(LL);
    int hop = get_hop(LL);
    int dis = get_distance(LL);
    int offset = hub_vertex * (hop_cst + 1) + hop;
    for (int x = hop; x <= hop_cst; ++ x) {
        if (hash_array[offset] > dis) {
            atomicMin(&hash_array[offset ++], dis);
        } else {
            break;
        }
    }
    in_L[hub_vertex] = 1;
    return;
}

// clean hash_table
__global__ void clear_hash_v3 (int V, int hop_cst, int vid, int *in_L, long long *L, long long *L_start, long long *L_end, int *hash_array) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < 0 || tid >= L_end[vid] - L_start[vid]) {
        return;
    }
    long long LL = L[L_start[vid] + tid]; // get the label need to get hash
    int hub_vertex = get_hub_vertex(LL);
    int hop = get_hop(LL);
    int offset = hub_vertex * (hop_cst + 1) + hop;
    for (int x = hop; x <= hop_cst; ++ x) {
        if (hash_array[offset] != (1 << 14)) {
            hash_array[offset ++] = (1 << 14);
        } else {
            break;
        }
        
    }
    in_L[hub_vertex] = 0;
    return;
}

__global__ void clean_check_v3 (int hop_cst, int vid, long long L_tot, int *in_L, long long *L, int *hash_array, int *mark) {
    long long tid = blockIdx.x * blockDim.x + threadIdx.x;

    if (tid < 0 || tid >= L_tot) {
        return;
    }

    if (mark[tid]) return;

    long long LL = L[tid];
    
    int st_vertex = (LL >> 37);
    int ed_vertex = (LL >> 13) & ((1 << 24) - 1);
    
    if (!in_L[st_vertex] || !in_L[ed_vertex]) {
        return;
    }

    int hop_now = (LL >> 10) & ((1 << 3) - 1);
    int dis = (LL) & ((1 << 10) - 1);
    st_vertex = st_vertex * (hop_cst + 1) + hop_now;
    ed_vertex = ed_vertex * (hop_cst + 1);

    for (int i = hop_now; i >= 0; -- i) {
        if (hash_array[st_vertex --] + hash_array[ed_vertex ++] <= dis) {
            mark[tid] = 1;
            return;
        }
    }

    return;
}

void gpu_clean_clear_v3 (hop_constrained_case_info_v2 *info_gpu) {
    L_tot_2 = 0;
    label_id.clear();
    free(L2_pos_2);
    free(L_size_2);

    hipFree(&info_gpu->L_start);
    hipFree(&info_gpu->L_end);
    hipFree(&info_gpu->L);
    hipFree(&info_gpu->L2);
    hipFree(&info_gpu->mark);
    hipFree(&info_gpu->hash_array);
    hipFree(&info_gpu->in_L);
}

void gpu_clean_init_v3 (graph_v_of_v<int> &input_graph, const vector<vector<hop_constrained_two_hop_label>> &use_L,
vector<vector<hop_constrained_two_hop_label>> &clean_L, vector<vector<long long>> &label_id_2to1, hop_constrained_case_info_v2 *info_gpu, Graph_pool<int> &graph_pool, int tc, int K) {
    gpu_clean_clear_v3 (info_gpu);
    
    // label_id
    int V = input_graph.size();

    vector<vector<hop_constrained_two_hop_label>> transfer_L;
    transfer_L.resize(V);

    // start get L
    // use_L is a label used to clean labels
    hop_constrained_two_hop_label temp;
    for (int i = 0; i < V; ++ i) {
        for (int j = 0; j < use_L[i].size(); ++ j) {
            temp.hub_vertex = use_L[i][j].hub_vertex;
            temp.hop = use_L[i][j].hop;
            temp.distance = use_L[i][j].distance;
            temp.parent_vertex = i;
            transfer_L[temp.parent_vertex].push_back(temp);
        }
    }

    vector<long long> L_flat;

    hipMallocManaged(&info_gpu->L_start, (long long) (V + 1) * sizeof(long long));
    hipMallocManaged(&info_gpu->L_end, (long long) (V + 1) * sizeof(long long));
    hipDeviceSynchronize();

    L_size_2 = (int*) malloc(sizeof(int) * V);

    long long point = 0;
    int x;
    for (int i = 0; i < V; ++i) {
        info_gpu->L_start[i] = point;
        int _size = transfer_L[i].size();
        for (int j = 0; j < _size; j++) {
            L_flat.push_back(get_label(transfer_L[i][j].hub_vertex, transfer_L[i][j].parent_vertex, 
                                       transfer_L[i][j].hop, transfer_L[i][j].distance));
        }
        point += _size;
        info_gpu->L_end[i] = point;
        L_size_2[i] = _size;
    }

    // get L
    hipMallocManaged(&info_gpu->L, (long long) L_flat.size() * sizeof(long long));
    hipDeviceSynchronize();
    hipMemcpy(info_gpu->L, L_flat.data(), (long long) L_flat.size() * sizeof(long long), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    // end get L

    // start get L2
    // clean_L is the label that needs to be cleaned
    vector<vector<std::pair<hop_constrained_two_hop_label, int>>> transfer_L_v2;
    transfer_L_v2.resize(V);
    
    for (int i = 0; i < V; ++ i) {
        for (int j = 0; j < clean_L[i].size(); ++ j) {
            hop_constrained_two_hop_label temp;
            temp.hub_vertex = i;
            temp.hop = clean_L[i][j].hop;
            temp.distance = clean_L[i][j].distance;
            temp.parent_vertex = clean_L[i][j].hub_vertex;
            transfer_L_v2[clean_L[i][j].hub_vertex].push_back(std::make_pair(temp, label_id_2to1[i][j]));
        }
    }

    vector<std::pair<long long, int>> L_flat_v2;
    L_flat.clear();

    for (int i = 0; i < V; ++ i) {
        int _size = transfer_L_v2[i].size();
        for (int j = 0; j < _size; j++) {
            hop_constrained_two_hop_label ll = transfer_L_v2[i][j].first;
            L_flat_v2.push_back(std::make_pair(get_label(ll.hub_vertex, ll.parent_vertex, 
                                       ll.hop, ll.distance), transfer_L_v2[i][j].second));
        }
        L_tot_2 += _size;
    }
    
    // get L2
    stable_sort(L_flat_v2.begin(), L_flat_v2.end(), cmp_LL);
    L2_pos_2 = (int*) malloc(sizeof(int) * L_flat_v2.size());
    
    for (int i = 0; i <= V; ++ i) L2_pos_2[i] = 0;
    for (int i = 0; i < L_flat_v2.size(); ++ i) {
        int now_mn = min(get_hub_vertex(L_flat_v2[i].first), get_parent_vertex(L_flat_v2[i].first));
        L_flat.push_back(L_flat_v2[i].first);
        L2_pos_2[now_mn] = i;
        label_id.push_back(L_flat_v2[i].second);
    }
    for (int i = 0; i < V; ++ i) L2_pos_2[i] = L2_pos_2[i + 1];
    
    for (int i = V; i > 0; -- i) {
        if (L2_pos_2[i] == 0) {
            L2_pos_2[i] = L2_pos_2[i + 1];
        }
    }
    
    // for(int i = V - 1000; i < V; ++ i) printf("%d, ", L2_pos_2[i]);
    // for(int i = 0; i < V; ++ i) {
    //     // printf("%d ", L2_pos_2[i]);
    //     if (L2_pos_2[i] == -1) {
    //         L2_pos_2[i] = L2_pos_2[i + 1];
    //     }
    //     // L2_pos_2[i] = max(L2_pos_2[i], L2_pos_2[i - 1]);
    // }
    
    // for(int i = 0; i <= V; ++ i) {
    //     if (L2_pos_2[i] == -1) {
    //         puts("--------------------111111111111111111111111111 !!!!!");
    //     }
    // }
    // get L2
    hipMallocManaged(&info_gpu->L2, (long long) L_flat.size() * sizeof(long long));
    hipDeviceSynchronize();
    hipMemcpy(info_gpu->L2, L_flat.data(), (long long) L_flat.size() * sizeof(long long), hipMemcpyHostToDevice);
    hipDeviceSynchronize();

    hipMallocManaged(&info_gpu->mark, (long long) L_flat.size() * sizeof(int));
    hipMemset(info_gpu->mark, 0, (long long) sizeof(int) * L_flat.size());

    hipMallocManaged(&info_gpu->hash_array, (long long) V * sizeof(int) * (K + 1));
    hipDeviceSynchronize();

    for (long long i = 0; i < (long long) V * (K + 1); i++){
        info_gpu->hash_array[i] = (1 << 14);
    }
    hipDeviceSynchronize();

    hipMallocManaged(&info_gpu->in_L, (long long) V * sizeof(int));
    hipDeviceSynchronize();
}

void gpu_clean_v3 (graph_v_of_v<int> &input_graph, hop_constrained_case_info_v2 *info_gpu, 
vector<vector<hop_constrained_two_hop_label>> &res, vector<int> &check_tot, int thread_num) {
    int V = input_graph.size();
    int K = info_gpu->hop_cst;

    long long *L_start = info_gpu->L_start;
    long long *L_end = info_gpu->L_end;

    long long *L = info_gpu->L;
    long long *L2 = info_gpu->L2;
    int *in_L = info_gpu->in_L;

    int *mark = info_gpu->mark;
    int *hash_array = info_gpu->hash_array; // first dim size is V * (K + 1)

    int start_id = V, end_id, start_node_id, end_node_id;

    double tot_duration = 0.0, tot_duration_check = 0.0;
    
    auto begin_for = std::chrono::high_resolution_clock::now();
    for (int i = 0; i < V; ++ i) {
        if (L_size_2[i]) {
            auto begin = std::chrono::high_resolution_clock::now();

            get_hash_v3 <<< (L_size_2[i] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>>
            (V, K, i, in_L, L, L_start, L_end, hash_array);
            hipDeviceSynchronize();
            
            auto begin_check = std::chrono::high_resolution_clock::now();
            clean_check_v3 <<< (L2_pos_2[i] + 1 + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>> 
            (K, i, L2_pos_2[i] + 1, in_L, L2, hash_array, mark);
            hipDeviceSynchronize();
            auto end_check = std::chrono::high_resolution_clock::now();
            auto duration_check = std::chrono::duration_cast<std::chrono::nanoseconds>(end_check - begin_check).count() / 1e9;
            tot_duration_check += duration_check;

            clear_hash_v3 <<< (L_size_2[i] + THREADS_PER_BLOCK - 1) / THREADS_PER_BLOCK, THREADS_PER_BLOCK >>> 
            (V, K, i, in_L, L, L_start, L_end, hash_array);
            hipDeviceSynchronize();

            auto end = std::chrono::high_resolution_clock::now();
            auto duration = std::chrono::duration_cast<std::chrono::nanoseconds>(end - begin).count() / 1e9;
            tot_duration += duration;
            
            if (i % 1000 == 0) {
                // printf("%lf, %lf\n", tot_duration, tot_duration_check);
                tot_duration = 0;
                tot_duration_check = 0;
            }
        }
        
    }
    auto end_for = std::chrono::high_resolution_clock::now();
    auto duration_for = std::chrono::duration_cast<std::chrono::nanoseconds>(end_for - begin_for).count() / 1e9;
    
    // printf("\n%lf\n", duration_for);

    long long LL;
    // for (int i = 0; i < V; ++ i) {
    //     res[i].clear();
    // }

    for (long long i = 0; i < L_tot_2; ++ i) {
        if (info_gpu->mark[i]) {
            // label_id[i] = 0;
            check_tot[label_id[i]] = 0;
        }
    }

    return;

}